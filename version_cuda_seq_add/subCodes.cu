#include "hip/hip_runtime.h"


#include <stdio.h>
#include <string.h>
#include "auxGPU.h"
#include "Events.h"
#include "threads.h"
#include "subCodes.h"



// Añade Word al subcódigo
__host__ __device__ sub_Code_t* add_Word(sub_Code_t* sc, Word_t* p) {
  sc->p[sc->M] = p;
  sc->M++;
  return sc;
}

// Añade Word al subcódigo
__global__ void d_add_Word(sub_Code_t* sc, Word_t* p) {
  sc->p[sc->M] = p;
  sc->M++;
}


// Imprime en pantalla el subcódigo
__host__ __device__ void print_Sub_Code(sub_Code_t* sc) {
    for (int i = 0; i < sc->M; i++ ) {
        printf("%4d: ", i);
        printWordBin((Word_t*)(sc->p[i]));
    }
}


// Retorna si el subcódigo es Separating o no
__host__ __device__ int dd_is_Sub_Code_Separating( sub_Code_t* sc, Word_t* p) {
    int block, pos;
    int n_Events =0;
    int cont = 1;
    for (int i = 1; i <= sc->M-2; i++) {
        for (int j = i+1; j<= sc->M-1; j++) {
            for (int k = j+1; k<=sc->M; k++) {

                if (DEPUR>DEPUR_2) printf("(%d,%d,%d,p)\n", i-1, j-1, k-1);
                Event_t* ev = new_Event(sc->p[i-1], sc->p[j-1], sc->p[k-1], p);
                n_Events++;
                if ( is_Event_Separating(ev, &block, &pos) ) {
                    if (DEPUR>DEPUR_3) printf("Event i-j-k Separating. block: %d, pos: %d\n", block, primer_Uno(pos));
                } else {
                    if (DEPUR>DEPUR_3) printf("Event i-j-k no Separating");
                    return FALSE; 
                }
                del_Event(ev);

                if (DEPUR>DEPUR_2) printf("(%d,%d,%d,p)\n", i-1, k-1, j-1);
                ev = new_Event(sc->p[i-1], sc->p[k-1], sc->p[j-1], p);
                n_Events++;
                if ( is_Event_Separating(ev, &block, &pos) ) {
                    if (DEPUR>DEPUR_3) printf("Event i-k-j Separating. block: %d, pos: %d\n", block, primer_Uno(pos));
                } else {
                    if (DEPUR>DEPUR_3) printf("Event i-k-j no Separating");
                    return FALSE; 
                }
                del_Event(ev);

                if (DEPUR>DEPUR_2) printf("(%d,%d,%d,p)\n", j-1, k-1, i-1);
                ev = new_Event(sc->p[j-1], sc->p[k-1], sc->p[i-1], p);
                n_Events++;
                if ( is_Event_Separating(ev, &block, &pos) ) {
                    if (DEPUR>DEPUR_3) printf("Event j-k-i Separating. block: %d, pos: %d\n", block, primer_Uno(pos));
                } else {
                    if (DEPUR>DEPUR_3) printf("Event j-k-i no Separating");
                    return FALSE; 
                }
                del_Event(ev);

if (DEPUR>DEPUR_1) { printf("%3d/%3d: | ", cont-1, (cont-1)*3); cont++; }
if (DEPUR>DEPUR_1) { printf("(%d,%d,%d,p) ", i, j, k);}
if (DEPUR>DEPUR_1) { printf("(%d,%d,%d,p) ", i, k, j);}
if (DEPUR>DEPUR_1) { printf("(%d,%d,%d,p)      ", j, k, i);}

if (DEPUR>DEPUR_1) { printf("(%d,%d,%d,p) ", i-1, j-1, k-1);}
if (DEPUR>DEPUR_1) { printf("(%d,%d,%d,p) ", i-1, k-1, j-1);}
if (DEPUR>DEPUR_1) { printf("(%d,%d,%d,p)\n", j-1, k-1, i-1);}

            }
               
        }
    }

    int M = sc->M;
    int n_Events_teo = M*(M-1)*(M-2)/2;
    if (DEPUR>DEPUR_1) printf("n_Events: %d; teóricos: %d\n", n_Events, n_Events_teo);
    return TRUE;
}



// Retorna si el subcódigo es Separating o no
__global__ void d_is_Sub_Code_Separating(sub_Code_t*  d_sc, Word_t* d_p, int* d_esSep ) {
    printf("d_sc->M: %d\n", d_sc->M);
    printf("d_p->n: %d\n", d_p->n);
    printf("d_p->N: %d\n", d_p->N);
    //*d_esSep = 27;
    *d_esSep = dd_is_Sub_Code_Separating( d_sc, d_p);
    //*d_esSep = 27;

    if ( *d_esSep ) {
            add_Word(d_sc, d_p);
            printf("(GPU)M: %d\n", d_sc->M);
            
        } else {
            printf("(GPU)candidato no era Separating\n");
        }

}





__global__ void is_Sub_Code_Separating_kernel_old(sub_Code_t*  d_sc, Word_t* d_p, int* d_esSep) {
    int thread_n = blockDim.x * blockIdx.x + threadIdx.x;
    int M = d_sc->M;
    int i, j, k;

    int threads_totales = nchoose3(M);

    extern __shared__ int sep[];

    //__shared__ int sep[1024];

    int block, pos;
    int n_Events =0;

    get_ijk(M, thread_n, &i, &j, &k);
    printf("M: %2d, thread: %2d, i=%2d, j=%2d, k=%2d\n", M, thread_n, i, j, k);

    *d_esSep = TRUE;

    if (DEPUR>DEPUR_2) printf("(%d,%d,%d,p)\n", i-1, j-1, k-1);
    Event_t* ev = new_Event(d_sc->p[i-1], d_sc->p[j-1], d_sc->p[k-1], d_p);
    n_Events++;
    if ( is_Event_Separating(ev, &block, &pos) ) {
        if (DEPUR>DEPUR_3) printf("Event i-j-k Separating. block: %d, pos: %d\n", block, primer_Uno(pos));
    } else {
        if (DEPUR>DEPUR_3) printf("Event i-j-k no Separating");
        sep[thread_n]=FALSE;
        printf("ABORT");
        del_Event(ev);
        goto final;
    }
    del_Event(ev);

    if (DEPUR>DEPUR_2) printf("(%d,%d,%d,p)\n", i-1, k-1, j-1);
    ev = new_Event(d_sc->p[i-1], d_sc->p[k-1], d_sc->p[j-1], d_p);
    n_Events++;
    if ( is_Event_Separating(ev, &block, &pos) ) {
        if (DEPUR>DEPUR_3) printf("Event i-k-j Separating. block: %d, pos: %d\n", block, primer_Uno(pos));
    } else {
        if (DEPUR>DEPUR_3) printf("Event i-k-j no Separating");
        sep[thread_n]=FALSE;
        printf("ABORT");
        del_Event(ev);
        goto final;
    }
    del_Event(ev);

    if (DEPUR>DEPUR_2) printf("(%d,%d,%d,p)\n", j-1, k-1, i-1);
    ev = new_Event(d_sc->p[j-1], d_sc->p[k-1], d_sc->p[i-1], d_p);
    n_Events++;
    if ( is_Event_Separating(ev, &block, &pos) ) {
        if (DEPUR>DEPUR_3) printf("Event j-k-i Separating. block: %d, pos: %d\n", block, primer_Uno(pos));
    } else {
        if (DEPUR>DEPUR_3) printf("Event j-k-i no Separating");
        sep[thread_n]=FALSE;
        printf("ABORT");
        del_Event(ev);
        goto final;
    }
    del_Event(ev);
   
    
    sep[thread_n] = TRUE;
    final: __syncthreads();




    if (thread_n==0) {
        for (int i=0; i<threads_totales; i++) {
            if (!sep[i]) {
                *d_esSep = FALSE;
                printf("(GPU) No es Separating!!!");
                return;
            }
        }
        *d_esSep = TRUE;
        add_Word(d_sc, d_p);
        printf("(GPU)M: %d\n", d_sc->M);
    }

}



__global__ void is_Sub_Code_Separating_kernel_3x(sub_Code_t*  d_sc, Word_t* d_p, int* d_esSep) {

    int n_Event = blockDim.x * blockIdx.x + threadIdx.x;

    int M = d_sc->M;
    int i, j, k;

    int Events_totales = nchoose3(M);

    Event_t Event;
    Event_t* ev = &Event;

    __shared__ int sep[MAX_THREADS];
    

    sep[threadIdx.x] = TRUE;
    __syncthreads();

    int block, pos;

    while (n_Event < Events_totales) {

        get_ijk(M, n_Event, &i, &j, &k);
       // printf("M: %2d, thread: %2d, i=%2d, j=%2d, k=%2d\n", M, n_Event, i, j, k);

        //if (DEPUR>DEPUR_2) printf("(%d,%d,%d,p)\n", i-1, j-1, k-1);
        recargar_Event(ev, d_sc->p[i-1], d_sc->p[j-1], d_sc->p[k-1], d_p);
        if ( is_Event_Separating(ev, &block, &pos) ) {
            if (DEPUR>DEPUR_3) printf("Event i-j-k Separating. block: %d, pos: %d\n", block, primer_Uno(pos));
        } else {
            if (DEPUR>DEPUR_3) printf("Event i-j-k no Separating");
            sep[threadIdx.x]=FALSE;
            break;
        }

        //if (DEPUR>DEPUR_2) printf("(%d,%d,%d,p)\n", i-1, k-1, j-1);
        recargar_Event(ev, d_sc->p[i-1], d_sc->p[k-1], d_sc->p[j-1], d_p);
        if ( is_Event_Separating(ev, &block, &pos) ) {
            if (DEPUR>DEPUR_3) printf("Event i-k-j Separating. block: %d, pos: %d\n", block, primer_Uno(pos));
        } else {
            if (DEPUR>DEPUR_3) printf("Event i-k-j no Separating");
            sep[threadIdx.x]=FALSE;
            break;
        }

        //if (DEPUR>DEPUR_2) printf("(%d,%d,%d,p)\n", j-1, k-1, i-1);
        recargar_Event(ev, d_sc->p[j-1], d_sc->p[k-1], d_sc->p[i-1], d_p);
        if ( is_Event_Separating(ev, &block, &pos) ) {
            if (DEPUR>DEPUR_3) printf("Event j-k-i Separating. block: %d, pos: %d\n", block, primer_Uno(pos));
        } else {
            if (DEPUR>DEPUR_3) printf("Event j-k-i no Separating");
            sep[threadIdx.x]=FALSE;
            break;
        }
    
        n_Event += blockDim.x * gridDim.x;

    }  // while(Event_n<Events_totales)

    __syncthreads();



    for (int tam = blockDim.x; tam > 1; tam = (tam+1)/2) {
        int lim = tam/2;
        int inc = lim + (tam & 0x01);
        if (threadIdx.x < lim) {
            sep[threadIdx.x] = sep[threadIdx.x] && sep[threadIdx.x+inc];
        }
        __syncthreads();
    }
    if (threadIdx.x==0) {
        if (!sep[0]) {
            *d_esSep=FALSE;
        }
    }

} // is_Sub_Code_Separating_kernel






__global__ void is_Sub_Code_Separating_kernel(sub_Code_t*  d_sc, Word_t* d_p, int* d_esSep) {


    int n_Event = blockDim.x * blockIdx.x + threadIdx.x;
    int n_Event_base; 

    int M = d_sc->M;
    int i, j, k;

    int Events_totales = 3 * nchoose3(M);

    Event_t Event;
    Event_t* ev = &Event;

    __shared__ int sep[MAX_THREADS];
    

    sep[threadIdx.x] = TRUE;
    __syncthreads();

    int block, pos;
    char *Event_str;

    while (n_Event < Events_totales) {

        
        n_Event_base = n_Event / 3;
        get_ijk(M, n_Event_base, &i, &j, &k);
       // printf("M: %2d, thread: %2d, i=%2d, j=%2d, k=%2d\n", M, n_Event, i, j, k);

        int sub_Event = n_Event % 3;

        switch (sub_Event) {
            case 0: 
                //if (DEPUR>DEPUR_2) printf("(%d,%d,%d,p)\n", i-1, j-1, k-1);
                Event_str = (char*)"i-j-k";
                recargar_Event(ev, d_sc->p[i-1], d_sc->p[j-1], d_sc->p[k-1], d_p);
                break;
            case 1:
                //if (DEPUR>DEPUR_2) printf("(%d,%d,%d,p)\n", i-1, k-1, j-1);
                Event_str = (char*)"i-k-j";
                recargar_Event(ev, d_sc->p[i-1], d_sc->p[k-1], d_sc->p[j-1], d_p);       
                break;
            case 2:
                //if (DEPUR>DEPUR_2) printf("(%d,%d,%d,p)\n", j-1, k-1, i-1);
                Event_str = (char*)"j-k-i";
                recargar_Event(ev, d_sc->p[j-1], d_sc->p[k-1], d_sc->p[i-1], d_p);
                break;
        }

        if ( is_Event_Separating(ev, &block, &pos) ) {
            if (DEPUR>DEPUR_3) printf("Event %s Separating. block: %d, pos: %d\n", Event_str, block, primer_Uno(pos));
        } else {
            if (DEPUR>DEPUR_3) printf("Event %s no Separating", Event_str);
            sep[threadIdx.x]=FALSE;
            break;
        }

        n_Event += blockDim.x * gridDim.x;

    }  // while(Event_n<Events_totales)

    __syncthreads();



    for (int tam = blockDim.x; tam > 1; tam = (tam+1)/2) {
        int lim = tam/2;
        int inc = lim + (tam & 0x01);
        if (threadIdx.x < lim) {
            sep[threadIdx.x] = sep[threadIdx.x] && sep[threadIdx.x+inc];
        }
        __syncthreads();
    }
    if (threadIdx.x==0) {
        if (!sep[0]) {
            *d_esSep=FALSE;
        }
    }

} // is_Sub_Code_Separating_kernel