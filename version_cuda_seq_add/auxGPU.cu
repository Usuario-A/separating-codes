#include <stdio.h>
#include "auxGPU.h"


/* Control del último error CUDA */
__host__ void controlError(const char * comentario) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess){
        fprintf(stderr, "Fail in %s, (error: %s)!\n", comentario, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}    




























