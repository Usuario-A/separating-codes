#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "Words.h"

/*

Convenio: 
  - Los vectores se enumeran simbólicamente de izquierda a derecha:
     [0] [1] ... []
  - Los bits se agrupan de 32 en 32 (BITS_PER_TYPE_DEC) con la siguiente correspondencia:
     vector binario:  [0] [1] ... [31] [32] [33] ... [63]   
     vector decimal:  [       0      ] [        1       ]

*/

/* Pasa de cantidad de bits a cantidad de Words necesarias para contenerlos */
__host__ __device__ int nBits2nWords(int n){
  return (int) ceil( (double)n / BITS_PER_TYPE_DEC );
}

/* Genera una nueva Word de n bits*/
__host__ __device__ Word_t* new_Word(int n){
  Word_t* p = (Word_t*) malloc(sizeof(Word_t)); 
  p->n = n;
  p->N = nBits2nWords(n);
  p->v = (TYPE_DEC*) malloc(p->N*sizeof(TYPE_DEC));
  return p;
}

/* Libera memoria de variable de tipo Word_t */
__host__ __device__ Word_t* del_Word(Word_t* p){
  free(p->v);
  free(p);
  return p;
}


/* Imprime Word en formato decimal */
__host__ __device__ void printWordDec(Word_t* p) {
  for ( int i = 0; i<p->N; i++ ) {
    printf(FMT_UINT " ", p->v[i]);
  }
  printf("\n");
}


/* Imprime Word en formato binario */
__host__ __device__ void printWordBin(Word_t* p) {
  int n = p->n;
  int N = nBits2nWords(p->n);

  //TYPE_BIN cBin[n];
  TYPE_BIN* cBin = (TYPE_BIN*) malloc(sizeof(TYPE_BIN)*p->n);

  cDec2cBin (N, p->v, n, cBin);

  for ( int i=0; i < n%BITS_PER_TYPE_DEC; i++ ) {
    printf("%d",  ((cBin[i]!=0)?1:0) );
    if (n%BITS_PER_TYPE_DEC-i==5) printf(" ");   
    if (i==n%BITS_PER_TYPE_DEC-1) printf("  ");
  }
  
  for ( int i = n%BITS_PER_TYPE_DEC; i<n; i++ ) {
    printf("%d",  ((cBin[i]!=0)?1:0) );
    if (!((i-n%BITS_PER_TYPE_DEC+1)%BITS_PER_TYPE_DEC)) printf("  "); 
    else if (!((i-n%BITS_PER_TYPE_DEC+1)%4)) printf(" ");
    
  }
  printf("\n");
  free(cBin);

}


/* De vector binario (vector de n bits) a vector decimal (vector de N decimales), codificando la misma información.
 */
__host__ __device__ void cBin2cDec (int n, TYPE_BIN* cBin, int N, TYPE_DEC* cDec) {
 
  int cont = 0;
  int j_inic = 0;

  TYPE_DEC aux = 0;
  for ( int i = 0; i < n % BITS_PER_TYPE_DEC; i++ ) {
    aux = (aux<<1) | (cBin[cont++]!=0); 
    j_inic = 1;
  }
  cDec[0] = aux;

  for ( int j = 0; j < n / BITS_PER_TYPE_DEC; j++ ) {
    aux = 0;
    for ( int i = 0; i < BITS_PER_TYPE_DEC; i++ ) {
        aux = (aux<<1) | (cBin[cont++]!=0); 
      }
    cDec[j+j_inic] = aux;
  }

}





/* De Word decimal (vector de N decimales) a Word binaria (vector de n bits), codificando la misma información. 
*/
__host__ __device__ void cDec2cBin (int N, TYPE_DEC* cDec, int n, TYPE_BIN* cBin) {

  int cont = 0;
  int j_inic = 0;

  TYPE_DEC mascara = 1 << ( (n % BITS_PER_TYPE_DEC)-1 );
  for ( int i = 0; i < n % BITS_PER_TYPE_DEC; i++ ) {
    cBin[cont++] = (cDec[0] & mascara) != 0;
    mascara >>= 1;
    j_inic = 1;
  }

  for ( int j = j_inic; j <=  n / BITS_PER_TYPE_DEC; j++ ) {
    mascara = MSB_TYPE_DEC;
    for ( int i = 0; i < BITS_PER_TYPE_DEC; i++ ) {
      cBin[cont++] = (cDec[j] & mascara) != 0;
      mascara >>= 1;
    }
  }

}


// Devuelve la posición del primer dígito binario a 1 del decimal i, 
// comenzando desde su LSB
__host__ __device__ int primer_Uno(int i) {
  Word_t* p = new_Word(BITS_PER_TYPE_DEC);
  p->v[0]=i;
  //TYPE_BIN cBin[p->n];
  TYPE_BIN* cBin = (TYPE_BIN*) malloc(sizeof(TYPE_BIN)*p->n);
  cDec2cBin(p->N, p->v, p->n, cBin);
  for (int j=0; j<p->n; j++) {
    if (cBin[j]!=0) {
      return j;
    }
  }
  free(cBin);
  return -1;
}




/** genera una Word aleatoria binaria */
TYPE_BIN* rand_Word_Bin(int n, TYPE_BIN* vBin) {
  for ( int i = 0; i < n; i++ ) {
    vBin[i] = (rand()>RAND_MAX/2)? 1 : 0;
  }
  return vBin;
}

/* Genera una Word aleatoria decimal */
Word_t* rand_Word_Dec(int n) {
  Word_t* p = new_Word(n);
  TYPE_BIN vBin[p->n];
  cBin2cDec(p->n, rand_Word_Bin(p->n, vBin), nBits2nWords(p->n), p->v);
  return p;
}