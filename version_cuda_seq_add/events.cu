#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include "Events.h"

// Reserva espacio para un Event y referencia a las cuatro Words en cuestión
__host__ __device__ Event_t* new_Event(Word_t* a, Word_t* b, Word_t *c, Word_t *d) {
  Event_t* ev = (Event_t*) malloc(sizeof(Event_t));
  ev->a = a;
  ev->b = b;
  ev->c = c;
  ev->d = d;
  return ev;
}


// Idem que new_Event, pero sin reservar espacio de nuevo: aprovecha el ya reservado
__host__ __device__ Event_t* recargar_Event(Event_t* ev, Word_t* a, Word_t* b, Word_t *c, Word_t *d) {
  ev->a = a;
  ev->b = b;
  ev->c = c;
  ev->d = d;
  return ev;
}

// Libera el espacio destinado a un Event
__host__ __device__ Event_t* del_Event(Event_t* ev) {
  free(ev);
  return ev;
}


// Comprueba si cuatro números decimales (32 bits cada uno) pueden considerarse "Separatings"
__host__ __device__ int is_Event_Word_Separating(TYPE_DEC a, TYPE_DEC b, TYPE_DEC c, TYPE_DEC d, int *pos) {
  if (DEPUR>DEPUR_5) printf("is_Word_Separating? a=" FMT_UINT ", b=" FMT_UINT ", c=" FMT_UINT ", d=" FMT_UINT "\n", a, b, c, d);
  return (*pos =  (a^c)&(b^d)&(~(a^b))) != 0;
}

// Comprueba que un Event es Separating (basándose en la separabilidad de cada entero de los
// que se componga)
__host__ __device__ int is_Event_Separating(Event_t* ev, int *bloque, int *pos) {
  int N = nBits2nWords( ev->a->n );
  for (int i = 0; i < N; i++ ) {
    if ( is_Event_Word_Separating(ev->a->v[i], ev->b->v[i], ev->c->v[i], ev->d->v[i], pos) ) {
      *bloque = i;
      if (DEPUR>DEPUR_5) { print_Event_Bin(ev); printf("\n"); }
      return TRUE;
    }
  }
  return FALSE;
}

// Imprime en pantalla un Event en formato binario
__host__ __device__ void print_Event_Bin(Event_t* ev) {
  printWordBin(ev->a);
  printWordBin(ev->b);
  printWordBin(ev->c);
  printWordBin(ev->d);
}

// Imprime en pantalla un Event en formato decimal
__host__ __device__ void print_Event_Dec(Event_t* ev) {
  printWordDec(ev->a);
  printWordDec(ev->b);
  printWordDec(ev->c);
  printWordDec(ev->d);
}


