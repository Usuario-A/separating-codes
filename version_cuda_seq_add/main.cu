#include <stdio.h>

#include <hip/hip_runtime.h>

#include "general.h"
#include "Words.h"
#include "Events.h"
#include "subCodes.h"
#include "auxGPU.h"
#include "threads.h"
#include "results.h"




int main(void) {

    

    struct parametros datos;
    struct info results;

    unsigned int seed;
    seed = time(NULL);
    seed = 1;
    srand(seed);
    datos.seed = seed;


    // Control de MAX_THREADS
    int d;
    hipDeviceProp_t prop;
    CUDA_CALL( hipGetDevice(&d) );
    CUDA_CALL( hipGetDeviceProperties(&prop, d) );
    if ( prop.maxThreadsPerBlock < MAX_THREADS ) {
        printf("Maxim number of threads por block (%d) is lower than needed (%d)", 
        prop.maxThreadsPerBlock, MAX_THREADS);
        exit(EXIT_FAILURE);
    }

    printf("name: %s\nmultiProcessorCount: %d\nmaxThreadsPerMultiprocessor: %d\nmaxThreadsPerBlock: %d\nmaxThreadsDim: (%d,%d,%d)\n", prop.name, prop.multiProcessorCount, prop.maxThreadsPerMultiProcessor, prop.maxThreadsPerBlock, prop.maxThreadsDim[0], 
    prop.maxThreadsDim[1],prop.maxThreadsDim[2]);


    strcpy(datos.name, prop.name);
    datos.multiProcessorCount = prop.multiProcessorCount;
    datos.maxThreadsPerMultiProcessor = prop.maxThreadsPerMultiProcessor;
    datos.maxThreadsPerBlock = prop.maxThreadsPerBlock;
    datos.maxThreadsDim[0] = prop.maxThreadsDim[0];
    datos.maxThreadsDim[1] = prop.maxThreadsDim[1];
    datos.maxThreadsDim[2] = prop.maxThreadsDim[2];



    //  Code word length
    int n = 128;

    datos.n = n;

    // Desired number of words
    int M = 300;
    datos.M = M;


    // Event inicial con las primeras 4 Words candiato
    Event_t* ev = new_Event(rand_Word_Dec(n), rand_Word_Dec(n), rand_Word_Dec(n), rand_Word_Dec(n));

    // Verificación de separabilidad de primer Event
    int block, pos;
    while (!is_Event_Separating(ev, &block, &pos)) {
        printf("Initial Event is not Separating\n");
        del_Event(ev);
        ev = new_Event(rand_Word_Dec(n), rand_Word_Dec(n), rand_Word_Dec(n), rand_Word_Dec(n));
    }
    printf("Initial Event is already Separating. block: %d, pos=%d\n", block, primer_Uno(pos));

    // subcódigo local
    sub_Code_t sc;
    sc.n = n;
    sc.N = nBits2nWords(n);
    sc.M = 0;

    // add first four words ódigo del primer Event Separating
    add_Word(&sc, ev->a);
    add_Word(&sc, ev->b);
    add_Word(&sc, ev->c);
    add_Word(&sc, ev->d);

    del_Event(ev);
    
    printf("*************\n");
    print_Sub_Code(&sc);
    printf("*************\n");


    // reflejo inicial del subCode del en la GPU
    sub_Code_t sc_aux;
    sc_aux.M = sc.M;
    sc_aux.N = sc.N;
    sc_aux.n = sc.n;

    
    // Cara en GPU de las Words del subcódigio inicial de 4 Words
    for (int i = 0; i < sc.M; i++) {
        // Carga del vector
        TYPE_DEC* d_v;
        printf("sizeof(TYPE_DEC)*sc.p[%d]=%d", i, sizeof(TYPE_DEC)*sc.p[i]->N);
        CUDA_CALL( hipMalloc((void**)&d_v, sizeof(TYPE_DEC)*sc.p[i]->N) );        
        CUDA_CALL( hipMemcpy(d_v, sc.p[i]->v, sizeof(TYPE_DEC)*sc.p[i]->N, hipMemcpyHostToDevice ) );
        
        // Carga de la Word
        Word_t* d_p;
        CUDA_CALL( hipMalloc((void**)&d_p, sizeof(Word_t)) );
        Word_t* p = new_Word(sc.p[i]->n);
        p->v = d_v;
        CUDA_CALL( hipMemcpy(d_p, p, sizeof(Word_t), hipMemcpyHostToDevice) );
        
        sc_aux.p[i] = d_p;
    }
    // Carga del subcódigo completamente inicializado
    sub_Code_t* d_sc;
    CUDA_CALL( hipMalloc((void**)&d_sc, sizeof(sub_Code_t)) );
    CUDA_CALL( hipMemcpy(d_sc, &sc_aux, sizeof(sub_Code_t), hipMemcpyHostToDevice) );
    


    // Numero de Words aleatorias generadas no Separatings
    int n_no_sep=0;
    // Testigo local de separabilidad o no de Word candidato
    int esSep;
    // Testigo en GPU de separabilidad o no de Word candidato
    int* d_esSep = NULL;
    CUDA_CALL( hipMalloc((void**)&d_esSep, sizeof(int)) );
    

    int cierto = TRUE;
    int* TRUE_ptr = &cierto;


    hipEvent_t start_iteracion, stop_iteracion;
    CUDA_CALL( hipEventCreate(&start_iteracion) );
    CUDA_CALL( hipEventCreate(&stop_iteracion) );


    hipEvent_t start_pre_kernel, stop_pre_kernel;
    CUDA_CALL( hipEventCreate(&start_pre_kernel) );
    CUDA_CALL( hipEventCreate(&stop_pre_kernel) );

    hipEvent_t start_kernel, stop_kernel;
    CUDA_CALL( hipEventCreate(&start_kernel) );
    CUDA_CALL( hipEventCreate(&stop_kernel) );


    hipEvent_t start_post_kernel, stop_post_kernel;
    CUDA_CALL( hipEventCreate(&start_post_kernel) );
    CUDA_CALL( hipEventCreate(&stop_post_kernel) );


    hipEvent_t t_inic, t_fin;
    CUDA_CALL( hipEventCreate(&t_inic) );
    CUDA_CALL( hipEventCreate(&t_fin) );

    CUDA_CALL( hipEventRecord(t_inic, 0) ); 

    float t_Word_Code_acum = 0.;

    // Generación de Words Separatings
    int cont = sc.M;
    int inic = cont + 1; 
    int i;
    for (i = 0; cont < M; i++) {
    
        /**/CUDA_CALL( hipEventRecord(start_iteracion, 0) );
        /**/CUDA_CALL( hipEventRecord(start_pre_kernel, 0) );


        // Generación de Word candidato
        Word_t* candidato = rand_Word_Dec(n);
        
        // Carga en GPU de Word candidato
        TYPE_DEC* d_v;
        CUDA_CALL( hipMalloc((void**)&d_v, sizeof(TYPE_DEC)*candidato->N) );    
        CUDA_CALL( hipMemcpy(d_v, candidato->v, sizeof(TYPE_DEC)*candidato->N, hipMemcpyHostToDevice) );
        

        // Uso de candidato auxiliar para cargar en host el vector candidato que ya reside en GPU
        Word_t* candidato_aux = new_Word(n);
        free(candidato_aux->v);
        candidato_aux->v = d_v;      

        // Reserva de espacio de Word candidato en GPU
        Word_t* d_candidato;
        CUDA_CALL( hipMalloc( (void**)&d_candidato, sizeof(Word_t) ) );  
        
        // Carga de candidato host en GPU
        CUDA_CALL( hipMemcpy(d_candidato, candidato_aux, sizeof(Word_t), hipMemcpyHostToDevice) );
        free(candidato_aux);


        // recarga de testigo de separabilidad en GPU a TRUE
        CUDA_CALL( hipMemcpy(d_esSep, TRUE_ptr, sizeof(int), hipMemcpyHostToDevice) );
        

        // Parametros arranque de kernel        
        int n_Events = 3 * nchoose3(sc.M);
        int threads_por_block = MIN(n_Events, MAX_THREADS);
        //threads_por_block = MIN(n_Events, 128);
        int blocks = ( n_Events + threads_por_block - 1 ) / threads_por_block;
        //blocks = MIN(blocks, 800);

        /**/results.n_Events[i] = n_Events;
        /**/results.blocks[i] = blocks;
        /**/results.threads[i] = threads_por_block;


        printf("n=%d, M=%d, Total Events=%d, <<<%dx%d>>>=%d\n", n, sc.M, n_Events, blocks, threads_por_block, blocks*threads_por_block);

        /**/CUDA_CALL( hipEventRecord(stop_pre_kernel, 0) );
        /**/CUDA_CALL( hipEventSynchronize(stop_pre_kernel) );
        /**/float t_pre_kernel; 
        /**/CUDA_CALL( hipEventElapsedTime(&t_pre_kernel, start_pre_kernel, stop_pre_kernel) );
        
        /**/CUDA_CALL( hipEventRecord(start_kernel, 0) );

        // Arranque de kernel
        is_Sub_Code_Separating_kernel<<<blocks,threads_por_block>>>(d_sc, d_candidato, d_esSep) ;
        
        /**/CUDA_CALL( hipEventRecord(stop_kernel, 0) );
        /**/CUDA_CALL( hipEventSynchronize(stop_kernel) );
        /**/float t_kernel; 
        /**/CUDA_CALL( hipEventElapsedTime(&t_kernel, start_kernel, stop_kernel) );
        
        /**/CUDA_CALL( hipEventRecord(start_post_kernel, 0) );

        // Carga de testigo de separabilidad de Word candidato en host
        CUDA_CALL( hipMemcpy(&esSep, d_esSep, sizeof(int), hipMemcpyDeviceToHost) );
        //printf("Candidate Separating? %s\n", ((esSep)?"TRUE":"FALSE"));

        /**/float t_post_kernel; 

        if (esSep) {
            add_Word(&sc, candidato);
            printf("(Host)M: %d\n", sc.M);

            // kernel que añade Word candidato como Word Separating
            d_add_Word<<<1,1>>>(d_sc, d_candidato);
            
            /**/CUDA_CALL( hipEventRecord(stop_post_kernel, 0) );
            /**/CUDA_CALL( hipEventSynchronize(stop_post_kernel) );
            /**/CUDA_CALL( hipEventElapsedTime(&t_post_kernel, start_post_kernel, stop_post_kernel) );

            /**/results.t_Word_Code[cont] = t_Word_Code_acum + t_pre_kernel + t_kernel + t_post_kernel;
            cont++; 
            t_Word_Code_acum = 0;
        
        } else {
            del_Word(candidato);
            CUDA_CALL( hipFree(d_v) );
            CUDA_CALL( hipFree(d_candidato) );
            printf("(host)candidate not Separating\n");
            n_no_sep++;

            CUDA_CALL( hipEventRecord(stop_post_kernel, 0) );
            CUDA_CALL( hipEventSynchronize(stop_post_kernel) );
            CUDA_CALL( hipEventElapsedTime(&t_post_kernel, start_post_kernel, stop_post_kernel) );

            t_Word_Code_acum += t_pre_kernel + t_kernel + t_post_kernel;
        }


        CUDA_CALL( hipEventRecord(stop_iteracion, 0) );
        CUDA_CALL( hipEventSynchronize(stop_iteracion) );

        float t_iteracion; 
        CUDA_CALL( hipEventElapsedTime(&t_iteracion, start_iteracion, stop_iteracion) );
        

        results.Word_Code[i] = cont;
        results.Word[i] = inic + i;

        results.t_pre_kernel[i]  = t_pre_kernel;
        results.t_kernel[i]      = t_kernel;
        results.t_post_kernel[i] = t_post_kernel;

        results.t_Word[i] = t_pre_kernel + t_kernel + t_post_kernel;

        printf("i=%d\n", i);
        results.t_iteracion[i] = t_iteracion;

        
    } // for (int cont = 0;


    CUDA_CALL( hipEventRecord(t_fin, 0) );
    CUDA_CALL( hipEventSynchronize(t_fin) );
    float tx;
    CUDA_CALL( hipEventElapsedTime(&tx, t_inic, t_fin) );

    printf("Simulation time:  %f s\n", tx/1000.);

    printf("Discarded: %d\n", n_no_sep);


    char fichero[256];


    sprintf(fichero, "cod%d.m",n);
    printf("Saving Code in %s\n", fichero);
    save_Code(&sc, fichero);

    
    sprintf(fichero, "results_cod%d.m",n);
    printf("Saving results in %s\n", fichero);
    guardar_results(fichero, &datos, &results, i, cont);

    /************************************/

    CUDA_CALL( hipDeviceReset() );
    
    printf("End\n");
    return 0;
}
