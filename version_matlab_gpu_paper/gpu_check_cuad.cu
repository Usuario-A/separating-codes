
#include <hip/hip_runtime.h>
__global__ void gpu_check_cuad(const int *Codigo,
                                 const int *quadr,
                                 int m,
                                 int n,
                                 int nquad,
                                 int * es_sep)
{
int tid=threadIdx.x+blockDim.x*blockIdx.x;
int i1, i2, i3, i4,j, v1, v2, v3, v4, iaux1;
while(tid<nquad)
   {
    iaux1=0;
    i1=quadr[tid];
    i2=quadr[tid+nquad];
	i3=quadr[tid+2*nquad];
    i4=quadr[tid+3*nquad];
	   for(j=0;j<n;j++) 
	     {v1=Codigo[i1+m*j];
		  v2=Codigo[i2+m*j];
		  v3=Codigo[i3+m*j];
          v4=Codigo[i4+m*j];
        
		  if ((v1==v2)&&(v3==v4)&&(v1!=v3))
		      iaux1=1;
		  
          }
        es_sep[tid]=iaux1;
        tid+=blockDim.x*gridDim.x;		  
       }

}